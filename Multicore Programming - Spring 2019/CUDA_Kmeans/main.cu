#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>

#define MAX_GRID_SIZE 65535
#define MAX_BLOCK_SIZE_X 32              // which points
#define MAX_BLOCK_SIZE_Y 4               // which cluster/dimension (first one for clustering, second one for computing centroids)
#define MAX_BLOCK_SIZE_Z 8               // which dimension
#define MAX_THREADS_PER_BLOCK 1024

#define FILE_NAME "points.txt"

using namespace std;

__global__ void kmeans(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int *converged);
__global__ void initialize(int *clusters, int n, int k);
__device__ float euclidean_dist(float *p1, float *p2, int d);
vector<string> split(string str, char delimiter);
__global__ void assign_cluster(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int *converged);
__global__ void compute_centroids(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int reset);

int main(){
        /* Read FILE_NAME with the following format:
           First line specifies n: the number of points, d: the dimension of the points, and k: the number of clusters, in the same order.
           The following n lines will contain the value of the nodes in each d dimension in each line. 
         */
        int n, d, k; // parameters
        float *points; // value of the points
        int *clusters; // each point's cluster a.k.a the final result
        int *converged; // whether the program is finished yet or not

        // open the file
        ifstream file (FILE_NAME);
        if (!file.is_open()){
                cout << "couldn't open file" << endl;
                return 0;
        }
        // read the parameters
        string line;
        getline(file, line);
        vector<string> params = split(line, ' ');
        n = stoi(params[0]);
        d = stoi(params[1]);
        k = stoi(params[2]);
        // read the points
        points = (float *)malloc(n * d * sizeof(float));
        for (int i = 0; i < n; i++){
                getline(file, line);
                vector<string> values = split(line, ' ');
                for (int j = 0; j < d; j++){
                        points[i * d + j] = stof(values[j]);
                }
        }
        file.close();

        /* Allocate memory on host for the clusters and the convergence state. */
        clusters = (int *) malloc(n * sizeof(int));
        converged = (int *) malloc(sizeof(int));

        /* Allocate memory on device */
        float *points_dev, *centroids_dev;
        int *clusters_dev, *clusters_size_dev;
        int *converged_dev;

        hipMalloc((void **) &points_dev, n * d * sizeof(float));
        hipMalloc((void **) &centroids_dev, k * d * sizeof(float));
        hipMalloc((void **) &clusters_dev, n * sizeof(int));
        hipMalloc((void **) &clusters_size_dev, k * sizeof(int));
        hipMalloc((void **) &converged_dev, sizeof(int));

        /* Copy the points from host to device */
        hipMemcpy(points_dev, points, n * d * sizeof(float), hipMemcpyHostToDevice);

        /* Compute the block size and grid size for initialization */
        int block_size = (n > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : n;
        int temp_grid_size = (n + block_size - 1) / block_size;
        int grid_size = (temp_grid_size > MAX_GRID_SIZE) ? MAX_GRID_SIZE : temp_grid_size;

        /* Initialize the clusters */
        initialize<<<grid_size, block_size>>>(clusters_dev, n, k);

        /* Compute the block size for launching the 'compute_centroids' kernel */
        int cc_block_size_x = (n > MAX_BLOCK_SIZE_X) ? MAX_BLOCK_SIZE_X : n;
        int cc_block_size_y = (d > MAX_BLOCK_SIZE_Y * MAX_BLOCK_SIZE_Z) ? (MAX_BLOCK_SIZE_Y * MAX_BLOCK_SIZE_Z) : d;

        /* Compute the block size for launching the 'assign_cluster' kernel */
        int ac_block_size_x = (n > MAX_BLOCK_SIZE_X) ? MAX_BLOCK_SIZE_X : n;
        int ac_block_size_y = (k > MAX_BLOCK_SIZE_Y) ? MAX_BLOCK_SIZE_Y : k;
        int ac_block_size_z = (d > MAX_BLOCK_SIZE_Z) ? MAX_BLOCK_SIZE_Z : d;

        /* Final computation of sizes */
        dim3 cc_block_dim(cc_block_size_x, cc_block_size_y);
        dim3 ac_block_dim(ac_block_size_x, ac_block_size_y, ac_block_size_z);

        /* because the first time, there is no need to reset centroids and clusters_size */
        int reset = 0;
        hipDeviceSynchronize();
        while (1){
                // initialize the converged flag
                memset(converged, 0, sizeof(int));
                hipMemcpy(converged_dev, converged, sizeof(int), hipMemcpyHostToDevice);

                // start the first kernel
                compute_centroids<<<grid_size, cc_block_dim>>>(points_dev, centroids_dev, clusters_dev, clusters_size_dev, n, d, k, reset);
                hipDeviceSynchronize();

                // start the second kernel 
                assign_cluster<<<grid_size, ac_block_dim>>>(points_dev, centroids_dev, clusters_dev, clusters_size_dev, n, d, k, converged_dev);
                hipDeviceSynchronize();

                // check converged flag
                hipMemcpy(converged, converged_dev, sizeof(int), hipMemcpyDeviceToHost);
                if (*converged == 0)
                        break;
                reset = 1;
        }

        /* Copy the results back from device to host */
        hipMemcpy(clusters, clusters_dev, n * sizeof(int), hipMemcpyDeviceToHost);

        /* do whatever you want with the results :)) */

        /* Free the memory */
        hipFree(points_dev);
        hipFree(centroids_dev);
        hipFree(clusters_dev);
        hipFree(clusters_size_dev);
        hipFree(converged_dev);
        free(points);
        free(clusters);
        free(converged);
}

vector<string>
split(string str, char delimiter){
        vector<string> internal;
        string word = "";
        for (int i = 0; i < str.size(); i++){
                char letter = str[i];
                if (letter == delimiter || letter == '\n'){
                        internal.push_back(word);
                        word = "";
                } else {
                        word += letter;
                }
        }
        internal.push_back(word);

        return internal;
}

__device__ float
euclidean_dist(float *p1, float *p2, int d){
        float dist = 0;
        for (int i = 0; i < d; i++){
                float dif = p1[i] - p2[i];
                dist += dif * dif;
        }
        return dist;

}

__global__ void
initialize(int *clusters, int n, int k){
        int start = threadIdx.x, stride = blockDim.x * gridDim.x;
        for (int i = start; i < n; i += stride){
                clusters[i] = i % k;
        }
}

__global__ void
compute_centroids(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int reset){
        int t_point = threadIdx.x, p_stride = blockDim.x * gridDim.x;
        int t_dim = threadIdx.y, d_stride = blockDim.y;

        /* reset centroids and their sizes */
        if (reset){
                if (threadIdx.x < k){
                        clusters_size[threadIdx.x] = 0;
                        for (int i = t_dim; i < d; i += d_stride){
                                centroids[threadIdx.x * d + i] = 0.0;
                        }
                }
        }
        __syncthreads();

        /* Each thread handles one (or a few) dimensions of one (or a few) points  */
        for (int i = t_point; i < n; i += p_stride){                                     // one (or a few) points
                int t_cluster = clusters[i];                                             // this point's cluster
                clusters_size[t_cluster]++;                                              // increase the size of this cluster
                for (int j = t_dim; j < d; j += d_stride){                               // one (or a few) dimensions
                        centroids[t_cluster * d + j] += points[i * d + j];               // this point's value added to this cluster's value
                }
        }
}


/* This function assigns clusters to points based on centroids */
__global__ void
assign_cluster(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int *converged){
        int t_point = threadIdx.x, t_cluster = threadIdx.y, t_dim = threadIdx.z;
        int p_stride = blockDim.x * gridDim.x, d_stride = blockDim.z;
        int t_converged = 0;

        __shared__ float tmp_points[MAX_BLOCK_SIZE_X * MAX_BLOCK_SIZE_Z];               // copying the points to shared memory
        __shared__ int tmp_clusters_size[MAX_BLOCK_SIZE_Y];
        __shared__ float distances[MAX_BLOCK_SIZE_X * MAX_BLOCK_SIZE_Y];                // the distance coresponding to each <point, centroid> pair
        __shared__ int block_converged;                                                 // flag indicating whether this block has converged or not

        tmp_clusters_size[t_cluster] = clusters_size[t_cluster];                        // copy the cluster sizes to shared memory 
        __syncthreads();

        /* Each thread handles one (or a few) dimensions of one (or a few) points next to one clusters */
        for (int i = t_point; i < n; i += p_stride){                                    // one (or a few) points

                /* Copy the points to shared memory */
                for (int j = t_dim; j < d; j += d_stride){
                        tmp_points[(i % p_stride) * d + j] = points[i * d + j];
                }
                __syncthreads();

                /* Calculate the <point, centroid> distances */
                for (int j = t_dim; j < n; j += d_stride){
                        float t_dist = (tmp_points[(i % p_stride) * d + j] - centroids[t_cluster * d + j]) / tmp_clusters_size[t_cluster] ;
                        distances[i * k + t_cluster] += t_dist * t_dist;
                }
                __syncthreads();

                /* assign the cluster */
                if (t_cluster == 0 && t_dim == 0){
                        /* compute minimum distance and coresponding cluster */
                        float min_dist = distances[i * k];
                        int curr_cluster = 0;
                        for (int l = 1; l < k; l++){
                                if (distances[i * k + l] < min_dist){
                                        min_dist = distances[i * k + l];
                                        curr_cluster = l;
                                }
                        }
                        /* set convergence flag and new (maybe) cluster */
                        if (clusters[i] != curr_cluster){
                                clusters[i] = curr_cluster;
                                t_converged = -1;
                        }
                }
                __syncthreads();
        }

        /* One thread checks the convergence of its assigned points */
        if (t_cluster == 0 && t_dim == 0){
                if (t_converged == -1){
                        block_converged = -1;
                }
        }
        __syncthreads();

        /* One thread from this block sets the convergence flag */
        if (t_point == 0 && t_cluster == 0 && t_dim == 0){
                if (block_converged == -1){
                        *converged = -1;
                }
        }
}


// THIS FUNCTION IS OLD!!!! 
__global__ void
kmeans(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int *converged){
        /* Compute the centroids */
        int start = threadIdx.x, stride = blockDim.x * gridDim.x;
        for (int i = start; i < n; i += stride){
                int cluster_i = clusters[i];
                clusters_size[cluster_i]++;
                for (int j = 0; j < d; j++){
                        centroids[cluster_i * d + j] += points[i * d + j];
                }
        }


        // TODO: GLOBAL SYNCHRONIZATION

        for (int i = start; i < k; i += stride){
                for (int j = 0; j < d; j++){
                        centroids[i * d + j] /= clusters_size[i];
                }
        }

        // TODO: GLOBAL SYNCHRONIZATION

        /* Compute each point's new cluster */
        for (int i = start; i < n; i += stride){
                // copy the current point to local memory to avoid constantly accessing global memory
                float *curr_point = (float *) malloc(d * sizeof(float));
                for (int l = 0; l < d; l++){
                        curr_point[l] = points[i * d + l];
                }
                float min_dif = euclidean_dist(curr_point, centroids, d), curr_dif = 0.0;
                int curr_cluster = 0;
                for (int j = 1; j < k; j++){
                        curr_dif = euclidean_dist(curr_point, centroids + j * d, d);
                        if (curr_dif < min_dif){
                                min_dif = curr_dif;
                                curr_cluster = j;
                        }
                }
                // TODO: a more efficient way of changing converged. 
                if (curr_cluster != clusters[i]){
                        *converged = -1;
                        clusters[i] = curr_cluster;
                }
        }
}
