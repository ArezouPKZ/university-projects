
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <random>

#define MAX_GRID_SIZE 4294967295
#define MAX_BLOCK_SIZE_X 32              // which points
#define MAX_BLOCK_SIZE_Y 8               // which cluster/dimension (first one for clustering, second one for computing centroids)
#define MAX_BLOCK_SIZE_Z 4               // which dimension
#define MAX_THREADS_PER_BLOCK 1024

#define MAX_ITER 32
#define FILE_NAME "points.txt"

using namespace std;

__global__ void initialize(int *clusters, int n, int k);
__global__ void compute_centroids(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int reset);
__global__ void assign_cluster(float *points, float *centroids, int *clusters, int *clusters_size, float *distances, int n, int d, int k, int *converged, int reset);

int main(){
        /* Read FILE_NAME with the following format:
           First line specifies n: the number of points, d: the dimension of the points, and k: the number of clusters, in the same order.
           The following n lines will contain the value of the nodes in each d dimension in each line.
         */

        int n = 1 << 20, d = 4, k = 8; // parameters
        float *points; // value of the points
        int *clusters; // each point's cluster a.k.a the final result
        int *converged; // whether the program is finished yet or not

        /* generate random numbers */
        points = (float *) malloc(n * d * sizeof(float));
        for (int i = 0; i < n; i++){
                for (int j = 0; j < d; j++){
                        float f = (float) rand() / RAND_MAX;
                        points[i * d + j] = f * 100.0;
                }
        }

        clusters = (int *) malloc(n * sizeof(int));
        converged = (int *) malloc(sizeof(int));

        /* Start overall timer */
        hipEvent_t start_overall, stop_overall;
        hipEventCreate(&start_overall);
        hipEventCreate(&stop_overall);
        hipEventRecord(start_overall, 0);

        /* Allocate memory on device */
        float *points_dev, *centroids_dev, *distances_dev;
        int *clusters_dev, *clusters_size_dev;
        int *converged_dev;

        hipMalloc((void **) &points_dev, n * d * sizeof(float));
        hipMalloc((void **) &centroids_dev, k * d * sizeof(float));
        hipMalloc((void **) &clusters_dev, n * sizeof(int));
        hipMalloc((void **) &clusters_size_dev, k * sizeof(int));
        hipMalloc((void **) &converged_dev, sizeof(int));
        hipMalloc((void **) &distances_dev, n * k * sizeof(float));

        /* Copy the points from host to device */
        hipMemcpy(points_dev, points, n * d * sizeof(float), hipMemcpyHostToDevice);

        /* Compute the block size and grid size for initialization */
        int block_size = (n > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : n;
        int temp_grid_size = (n + block_size - 1) / block_size;
        int grid_size = (temp_grid_size > MAX_GRID_SIZE) ? MAX_GRID_SIZE : temp_grid_size;

        /* Initialize the clusters */
        initialize<<<grid_size, block_size>>>(clusters_dev, n, k);

        /* Compute the block size for launching the 'compute_centroids' kernel */
        int cc_block_size_x = (n > MAX_BLOCK_SIZE_X) ? MAX_BLOCK_SIZE_X : n;
        int cc_block_size_y = (d > MAX_BLOCK_SIZE_Y * MAX_BLOCK_SIZE_Z) ? (MAX_BLOCK_SIZE_Y * MAX_BLOCK_SIZE_Z) : d;
        int cc_temp_grid_size = (n + cc_block_size_x - 1) / cc_block_size_x;
        int cc_grid_size = (cc_temp_grid_size > MAX_GRID_SIZE) ? MAX_GRID_SIZE : cc_temp_grid_size;

        /* Compute the block size for launching the 'assign_cluster' kernel */
        int ac_block_size_x = (n > MAX_BLOCK_SIZE_X) ? MAX_BLOCK_SIZE_X : n;
        int ac_block_size_y = (k > MAX_BLOCK_SIZE_Y) ? MAX_BLOCK_SIZE_Y : k;
        int ac_block_size_z = (d > MAX_BLOCK_SIZE_Z) ? MAX_BLOCK_SIZE_Z : d;
        int ac_temp_grid_size = (n + ac_block_size_x - 1) / ac_block_size_x;
        int ac_grid_size = (ac_temp_grid_size > MAX_GRID_SIZE) ? MAX_GRID_SIZE : ac_temp_grid_size;

        /* Final computation of sizes */
        dim3 cc_block_dim(cc_block_size_x, cc_block_size_y);
        dim3 ac_block_dim(ac_block_size_x, ac_block_size_y, ac_block_size_z);

        /* because the first time, there is no need to reset centroids and clusters_size */
        int reset = 0;
        hipDeviceSynchronize();

        /* Start execution timer */
        hipEvent_t start_exec, stop_exec;
        hipEventCreate(&start_exec);
        hipEventCreate(&stop_exec);
        hipEventRecord(start_exec, 0);

        int iter = 0;
        while (1){
                iter++;
                if (iter > MAX_ITER)
                        break;

                //initialize the convergence rate
                hipMemset(converged_dev, 0, sizeof(int));

                // start the first kernel
                compute_centroids<<<cc_grid_size, cc_block_dim>>>(points_dev, centroids_dev, clusters_dev, clusters_size_dev, n, d, k, reset);
                hipDeviceSynchronize();

                // start the second kernel
                hipMemset(distances_dev, 0, n * k * sizeof(float));
                assign_cluster<<<ac_grid_size, ac_block_dim>>>(points_dev, centroids_dev, clusters_dev, clusters_size_dev, distances_dev, n, d, k, converged_dev, reset);
                hipDeviceSynchronize();

                // check converged flag
                hipMemcpy(converged, converged_dev, sizeof(int), hipMemcpyDeviceToHost);
                if (*converged == 0)
                        break;
                reset = 1;
        }

        hipEventRecord(stop_exec, 0);
        hipEventSynchronize(stop_exec);

        /* Copy the results back from device to host */
        hipMemcpy(clusters, clusters_dev, n * sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(stop_overall, 0);
        hipEventSynchronize(stop_overall);

        /* do whatever you want with the results :)) */
        float overall_time, execution_time;
        hipEventElapsedTime(&overall_time, start_overall, stop_overall);
        hipEventElapsedTime(&execution_time, start_exec, stop_exec);
        cout << "n: " << n << "\tk: " << k << "\td: " << d << endl;
        cout << "converged after " << iter << " iterations" << endl;
        cout << "time for executing kmeans: " << execution_time << "ms" << endl;
        cout << "time for entire run (allocation, initialization, etc): " << overall_time << "ms" << endl;

        /* Free the memory */
        hipFree(points_dev);
        hipFree(centroids_dev);
        hipFree(clusters_dev);
        hipFree(clusters_size_dev);
        hipFree(converged_dev);
        free(points);
        free(clusters);
        free(converged);
        hipEventDestroy(start_overall);
        hipEventDestroy(stop_overall);
        hipEventDestroy(start_exec);
        hipEventDestroy(stop_exec);
}

__global__ void
initialize(int *clusters, int n, int k){
        int start = threadIdx.x, stride = blockDim.x * gridDim.x;
        for (int i = start; i < n; i += stride){
                clusters[i] = i % k;
        }
}

__global__ void
compute_centroids(float *points, float *centroids, int *clusters, int *clusters_size, int n, int d, int k, int reset){
        int t_point = threadIdx.x + blockDim.x * blockIdx.x, p_stride = blockDim.x * gridDim.x;
        int t_dim = threadIdx.y, d_stride = blockDim.y;

        /* reset centroids and their sizes */
        if (reset){
                if (threadIdx.x < k){
                        clusters_size[threadIdx.x] = 0;
                        for (int i = t_dim; i < d; i += d_stride){
                                centroids[threadIdx.x * d + i] = 0.0;
                        }
                }
        __syncthreads();
        }

        /* Each thread handles one (or a few) dimensions of one (or a few) points  */
        for (int i = t_point; i < n; i += p_stride){                                     // one (or a few) points
                int t_cluster = clusters[i];                                             // this point's cluster
                atomicAdd(clusters_size + t_cluster, 1);                                 // increase the size of this cluster
                for (int j = t_dim; j < d; j += d_stride){                               // one (or a few) dimensions
                        atomicAdd(centroids + t_cluster * d + j,  points[i * d + j]);    // this point's value added to this cluster's value
                }
        }
}

/* This function assigns clusters to points based on centroids */
__global__ void
assign_cluster(float *points, float *centroids, int *clusters, int *clusters_size, float* distances, int n, int d, int k, int *converged, int reset){
        int t_point = threadIdx.x + blockIdx.x * blockDim.x, t_cluster = threadIdx.y, t_dim = threadIdx.z;
        int p_stride = blockDim.x * gridDim.x, c_stride = blockDim.y, d_stride = blockDim.z;
        int t_converged = 0;
        __shared__ int block_converged;

        /* Each thread handles one (or a few) dimensions of one (or a few) points next to one (or a few) clusters */
        for (int i = t_point; i < n; i += p_stride){                                    // one (or a few) points
                for (int j = t_cluster; j < k; j += c_stride){                          // one (or a few) clusters
                        for (int l = t_dim; l < d; l += d_stride){                      // one (or a few) dimensions
                                /* Calculate the <point, centroid> distances */
                                float t_dist = points[i * d + l] - centroids[j * d + l] / clusters_size[j];
                                atomicAdd(distances + i * k + j, t_dist * t_dist);
                        }
                }
                __syncthreads();

                /* Calculate minimum distancd for this point and assign (maybe) new cluster. */
                if (t_cluster == 0 && t_dim == 0){
                        float min_distance = distances[i * k];
                        int min_cluster = 0;
                        for (int j = 1; j < k; j++){
                                if (distances[i * k + j] < min_distance){
                                        min_distance = distances[i * k + j];
                                        min_cluster = j;
                                }
                        }
                        if (min_cluster != clusters[i]){
                                clusters[i] = min_cluster;
                                t_converged = -1;
                        }
                }
        }

        /* handle convergence */
        if (t_cluster == 0 && t_dim == 0){
                if (t_converged == -1){
                        block_converged = -1;
                }
        }

        __syncthreads();
        if (t_point == 0 && t_cluster == 0 && t_dim == 0){
                if (block_converged == -1){
                        *converged = -1;
                        block_converged = 0;
                }
        }
}
