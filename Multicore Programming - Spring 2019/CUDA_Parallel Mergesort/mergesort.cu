#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string>
#include <cstdlib>
#include <vector>
#include <fstream>

using namespace std;

//void write_array_to_file(int *array, int dim, string file_name);
void print_array(int *array, int size);
__device__ void merge(int *array, int size, int *copy);
__global__ void mergesort(int *array, int size, int *temp, int level);

int main(){
        /* Initialize n here */
        int n = 1 << 10;

        /* Make a random array of size n */
        int *array;
        array = (int *)malloc(n * sizeof(int));
        for (int i = 0; i < n; i++){
                array[n - i] = i;
        }

        /* Allocate memory on device */
        int *array_dev, *copy_dev;
        hipMalloc((void **) &array_dev, n * sizeof(int));
        hipMalloc((void **) &copy_dev, n * sizeof(int));

        hipMemcpy(array_dev, array, n * sizeof(int), hipMemcpyHostToDevice);

        hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 4);
        mergesort<<<1, 1>>>(array_dev, n, copy_dev, 0);
        hipDeviceSynchronize();

        /* Copy results back to host */
        hipMemcpy(array, array_dev, n * sizeof(int), hipMemcpyDeviceToHost);

        print_array(array, n);

        hipFree(array_dev);
        hipFree(copy_dev);
        free(array);
}


void print_array(int *array, int size){
        for (int i = 0; i < size; i++)
                cout << array[i] << " ";
        cout << endl;
}

__device__
void merge(int *array, int size, int *copy){
        int left = 0, right = size / 2, c_ind = 0;

        for (c_ind = 0; left < size / 2 && right < size; c_ind++){
                if (array[left] < array[right]){
                        copy[c_ind] = array[left];
                        left++;
                } else {
                        copy[c_ind] = array[right];
                        right++;
                }
        }

        while (left < size / 2){
                copy[c_ind] = array[left];
                left++;
                c_ind++;
        }
        while (right < size){
                copy[c_ind] = array[right];
                right++;
                c_ind++;
        }

        for (int i = 0; i < size; i++)
                array[i] = copy[i];
}

__device__
void bubble_sort(int *array, int size){
        int sorted = 0, temp;
        while (!sorted) {
                sorted = 1;
                for (int i = 1; i < size; i++){
                        if (array[i - 1] > array[i]){
                                temp = array[i - 1];
                                array[i - 1] = array[i];
                                array[i] = temp;
                                sorted = 0;
                        }
                }
        }
}


__global__
void mergesort(int *array, int size, int *copy, int level){
        if (size < 2) {
                return;
        }
        if (level == 3){
                bubble_sort(array, size);
                return;
        }

        mergesort<<<1, 1>>>(array, size / 2, copy, level + 1);
        mergesort<<<1, 1>>>(array + size / 2, size - size / 2, copy, level + 1);

        hipDeviceSynchronize();

        merge(array, size, copy);
}
